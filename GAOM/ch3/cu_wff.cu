#include "hip/hip_runtime.h"
#include "cu_wff.h"

#define _USE_MATH_DEFINES
#include <math.h>

/*
  warp(32 threads) reduce is more efficient on post-kepler GPUs
  https://devblogs.nvidia.com/parallelforall/faster-parallel-reductions-kepler/
*/
__inline__ __device__
float warpReduceSum(float val) {
	for (int offset = warpSize / 2; offset > 0; offset /= 2)
		val += __shfl_down(val, offset);
	return val;
}

/* cufft Complex number scaling */
static	__device__ __host__
inline hipfftComplex ComplexScale(hipfftComplex a, float s)
{
	hipfftComplex c;
	c.x = s*a.x;
	c.y = s*a.y;
	return c;
}

/* cufft Complex number multiplication */
static __device__ __host__
inline hipfftComplex ComplexMul(hipfftComplex a, hipfftComplex b)
{
	hipfftComplex c;
	c.x = a.x * b.x - a.y * b.y;
	c.y = a.x * b.y + a.y * b.x;
	return c;
}

/*---------------------------------------------CUDA Kernels-------------------------------------------------*/
/*
PURPOSE:
1. Generate the xf and yf for analytically computation of the Gaussian Window in Fourier Domain
[yf xf]=meshgrid(-fix(nn/2):nn-fix(nn/2)-1,-fix(mm/2):mm-fix(mm/2)-1); mm, nn are padded height&width
NOTE:
Currently only even size in each dimension is supported
INPUTS:
width, height: width and height of the xf and yf matrices
OUTPUTS:
d_out_xf, d_out_yf: Generated meshgrid
*/
__global__
void gen_xf_yf_Kernel(hipfftReal *d_out_xf,
	hipfftReal *d_out_yf,
	int iWidth, int iHeight)
{
	int i = threadIdx.y + blockIdx.y * blockDim.y;
	int j = threadIdx.x + blockIdx.x * blockDim.x;

	int id = i*iWidth + j;

	int iHalfy = iHeight / 2;
	int iHalfx = iWidth / 2;

	if (i < iHeight && j < iWidth)
	{
		d_out_xf[id] = j - iHalfx;
		d_out_yf[id] = i - iHalfy;
	}
}

/*
PURPOSE:
Point-wise multiplication of two matrices of complex numbers
INPUT:
d_in_a, d_in_b: Two matrices to be multiplied
iSize: size of the matrices
OUTPUT:
d_out_c: The result after multiplication
*/
__global__
void complex_pointwise_multiplication_kernel(hipfftComplex *d_in_a,
	hipfftComplex *d_in_b,
	int iSize,
	hipfftComplex *d_out_c)
{
	for (int i = threadIdx.x + blockIdx.x*blockDim.x;
		i < iSize;
		i += blockDim.x*gridDim.x)
	{
		d_out_c[i] =
			ComplexScale(ComplexMul(d_in_a[i], d_in_b[i]), 1.0f / iSize);
	}
}

/*
PURPOSE:
Explicitly Compute the FFT of the Gaussian Window
Fg=exp(-(xf*2*pi/mm-wxt).^2/2*sigmax*sigmax - (yf*2*pi/nn-wyt).^2/2*sigmay*sigmay)*sn2;
INPUTS:
d_in_xf, d_in_yf: meshgrid in frequency domain
iPaddedWidth, iPaddedHeight: Padded Gaussian Window size
wxt, wyt: frequencies in integer intervals
wxi, wyi: steps
wxl, wyl: lower bound of the frequencies
sigmax, sigmay: sigma's in x&y directions
sn2: normalization params (norm2 = 1);
OUTPUTS:
d_out_Fg: Fg
*/
__global__
void compute_Fg_kernel(hipfftReal *d_in_xf,
	hipfftReal *d_in_yf,
	int iPaddedWidth, int iPaddedHeight,
	int wxt, int wyt, float wxi,
	float wyi, float wxl, float wyl,
	float sigmax, float sigmay,
	float sn2,
	hipfftComplex *d_out_Fg)
{
	hipfftReal rwxt = wxl + hipfftReal(wxt) * wxi;
	hipfftReal rwyt = wyl + hipfftReal(wyt) * wyi;

	for (int i = threadIdx.x + blockIdx.x * blockDim.x;
		i < iPaddedHeight*iPaddedWidth;
		i += blockDim.x * gridDim.x)
	{
		hipfftReal tempx = d_in_xf[i] * 2 * hipfftReal(M_PI) * (1.0f / iPaddedWidth) - rwxt;
		hipfftReal tempy = d_in_yf[i] * 2 * hipfftReal(M_PI) * (1.0f / iPaddedHeight) - rwyt;
		tempx = -tempx * tempx * 0.5f * sigmax * sigmax;
		tempy = -tempy * tempy * 0.5f * sigmay * sigmay;

		d_out_Fg[i].x = exp(tempx + tempy) * sn2;
		d_out_Fg[i].y = 0;
	}
}

/*
PURPOSE:
2. Do the fftshift on xf and yf to be coincide with the CUFFT's results
NOTE:
Currently only even size in each dimension is supported
INPUTS:
width, height: width and height of the xf and yf matrices
OUTPUTS:
d_out_xf, d_out_yf: In-place fft-shifted xf, yf

*/
__global__
void fftshift_xf_yf_kernel(hipfftReal *d_out_xf,
	hipfftReal *d_out_yf,
	int iWidth, int iHeight)
{
	int i = threadIdx.y + blockIdx.y * blockDim.y;
	int j = threadIdx.x + blockIdx.x * blockDim.x;

	int id = i*iWidth + j;

	int iHalfx = iWidth / 2;
	int iHalfy = iHeight / 2;
	int iSlice = iWidth * iHeight;

	int idQ13 = iSlice / 2 + iHalfx;
	int idQ24 = iSlice / 2 - iHalfx;

	hipfftReal Tempx, Tempy;

	if (j < iHalfx && i < iHalfy)
	{
		Tempx = d_out_xf[id];
		Tempy = d_out_yf[id];

		// First Quadrant
		d_out_xf[id] = d_out_xf[id + idQ13];
		d_out_yf[id] = d_out_yf[id + idQ13];

		// Third Quadrant
		d_out_xf[id + idQ13] = Tempx;
		d_out_yf[id + idQ13] = Tempy;
	}
	else if (j >= iHalfx && j < iWidth && i < iHalfy)
	{

		Tempx = d_out_xf[id];
		Tempy = d_out_yf[id];

		// Second Quadrant
		d_out_xf[id] = d_out_xf[id + idQ24];
		d_out_yf[id] = d_out_yf[id + idQ24];

		// Fourth Quadrant
		d_out_xf[id + idQ24] = Tempx;
		d_out_yf[id + idQ24] = Tempy;
	}
}

/*
PURPOSE:
Feed the input f into the Padded matrix m_d_fPadded
INPUTS:
d_in_f: The input fringe pattern
iWidth, iHeight: size of the d_f
iPaddedWidth, iPaddedHeight: FFT preferred size after padding
OUTPUTS:
d_out_fPadded: The padded d_f
*/
__global__
void feed_fPadded_kernel(hipfftComplex *d_in_f,
	hipfftComplex *d_out_fPadded,
	int iWidth, int iHeight,
	int iPaddedWidth, int iPaddedHeight)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	int idImg = y * iWidth + x;
	int idPadded = y * iPaddedWidth + x;

	if (y < iPaddedHeight && x < iPaddedWidth)
	{
		if (y < iHeight && x < iWidth)
		{
			d_out_fPadded[idPadded].x = d_in_f[idImg].x;
			d_out_fPadded[idPadded].y = d_in_f[idImg].y;
		}
		else
		{
			d_out_fPadded[idPadded].x = 0;
			d_out_fPadded[idPadded].y = 0;
		}
	}
}

/*
PURPOSE:
Calculate the threshold value for the WFF if it's not specified using Parallel Reduction Algorithm
thr = 6*sqrt(mean2(abs(f).^2)/3);
INPUTS:
d_in:	 type of hipfftComplex input array
size: size(width*height) of the in
OUTPUS:
d_out: 1-element device array
*/
__global__
void compute_WFF_threshold_kernel(hipfftComplex *d_in, float *d_out, int size)
{
	float sum = float(0);

	for (int i = threadIdx.x + blockIdx.x * blockDim.x;
		i < size;
		i += blockDim.x*gridDim.x)
	{
		float abs = hipCabsf(d_in[i]);
		sum += abs*abs;
	}

	sum = warpReduceSum(sum);

	if (threadIdx.x % warpSize == 0)
		atomicAdd(d_out, sum);
}

/*
PURPOSE:
Initialize all WFF related matrices to 0's
INPUTS:
iWidth, iHeight: size of the final results
OUTPUTS:
d_out_filtered:
*/
__global__
void init_WFF_matrices_kernel(hipfftComplex *d_out_filtered,
	int iWidth, int iHeight)
{
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int x = threadIdx.x + blockIdx.x * blockDim.x;

	int idImg = y * iWidth + x;

	if (y < iHeight && x < iWidth)
	{
		d_out_filtered[idImg].x = 0;
		d_out_filtered[idImg].y = 0;
	}
}

/*
PURPOSE:
Threshold the spectrum sf
INPUTS:
iWidth, iHeight: image size
iPaddedWidth, iPaddedHeight: Padded size
thr: the threshold
OUTPUTS:
d_out_sf: sf after threshold
*/
__global__
void threshold_sf_kernel(hipfftComplex *d_out_sf,
	int iWidth, int iHeight,
	int iPaddedWidth, int iPaddedHeight,
	float thr)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	int idPadded = y * iPaddedWidth + x;

	if (y < iPaddedHeight && x < iPaddedWidth)
	{
		if (y < iHeight && x < iWidth)
		{
			if (hipCabsf(d_out_sf[idPadded]) < thr)
			{
				d_out_sf[idPadded].x = 0;
				d_out_sf[idPadded].y = 0;
			}
		}
		else
		{
			d_out_sf[idPadded].x = 0;
			d_out_sf[idPadded].y = 0;
		}
	}
}

/*
PURPOSE:
Update the partial results im_d_filtered of each stream
INPUTS:
iWidth, iHeight: image size
iPaddedWidth, iPaddedHeight: Padded size
d_in_im_sf: spectrum of each stream
OUTPUTS:
d_out_im_filtered: filtered image after of each stream
*/
__global__
void update_WFF_partial_filtered_kernel(hipfftComplex *d_in_im_sf,
	int iWidth, int iHeight,
	int iPaddedWidth, int iPaddedHeight,
	hipfftComplex *d_out_im_filtered)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	int idPadded = y * iPaddedWidth + x;
	int idImg = y*iWidth + x;

	if (y < iHeight && x < iWidth)
	{
		d_out_im_filtered[idImg].x += d_in_im_sf[idPadded].x;
		d_out_im_filtered[idImg].y += d_in_im_sf[idPadded].y;
	}
}

/*
PURPOSE:
Update the final z.filtered
INPUTS:
d_in_im_filtered: the partial filtered results in each stream
imgSize: size of the fringe pattern
OUTPTS:
d_out_filtered: the final results
*/
__global__
void update_WFF_final_filtered_kernel(hipfftComplex *d_in_im_filtered,
	int imgSize,
	hipfftComplex *d_out_filtered)
{
	for (int i = threadIdx.x + blockIdx.x*blockDim.x;
		i < imgSize;
		i += blockDim.x * gridDim.x)
	{
		d_out_filtered[i].x += d_in_im_filtered[i].x;
		d_out_filtered[i].y += d_in_im_filtered[i].y;
	}
}

/*
PURPOSE:
Scale the final results
INPUTS:
d_out_filtered: the unscaled final results
imagSize: size of the fringe pattern
wxi,wyi: step size of the frequencies
OUTPUT:
d_out_filtered: scaled final results
*/
__global__
void scale_WFF_final_filtered_kernel(hipfftComplex *d_out_filtered,
	int imgSize,
	float wxi, float wyi)
{
	float factor = 0.25f * (1.0f / float(M_PI*M_PI)) * wxi * wyi;

	for (int i = threadIdx.x + blockIdx.x*blockDim.x;
		i < imgSize;
		i += blockDim.x * gridDim.x)
	{
		d_out_filtered[i].x *= factor;
		d_out_filtered[i].y *= factor;
	}
}

/*--------------------------------------------------WFT2 Implementations-----------------------------------------------*/
WFF2_CUDAF::WFF2_CUDAF(int iWidth, int iHeight, 
	WFT2_DeviceResultsF& z)
	: m_iWidth(iWidth)
	, m_iHeight(iHeight)
	, m_rThr(-1)
	, m_d_rThr(nullptr)
	, m_d_fPadded(nullptr)
	, m_d_xf(nullptr)
	, m_d_yf(nullptr)
	, im_d_Fg(nullptr)
	, im_d_filtered(nullptr)
	, m_fCoreTime(0)
	, m_fMemTime(0)
	, m_fPrecomputeTime(0)
{
	// Get the number of SMs on GPU 
	hipDeviceGetAttribute(&m_iSMs, hipDeviceAttributeMultiprocessorCount, 0);

	// WFF default parameters initialization
	m_rSigmaX = 10.0f;
	m_rWxl = -2.0f - 3.0f / m_rSigmaX;
	m_rWxi = 1.0f / m_rSigmaX;
	m_rWxh = 2.0f + 3.0f / m_rSigmaX;

	m_rSigmaY = 10.0f;
	m_rWyl = -2.0f - 3.0f / m_rSigmaY;
	m_rWyi = 1.0f / m_rSigmaY;
	m_rWyh = 2.0f + 3.0f / m_rSigmaY;

	// scale for window so that norm2 of the window is 1. 
	m_rGaussianNorm2 = sqrt(4 * float(M_PI)*m_rSigmaX*m_rSigmaY);
}

WFF2_CUDAF::WFF2_CUDAF(int iWidth, int iHeight,
	float rSigmaX, float rWxl, float rWxh, float rWxi,
	float rSigmaY, float rWyl, float rWyh, float rWyi,
	float rThr,
	WFT2_DeviceResultsF &z)
	: m_iWidth(iWidth)
	, m_iHeight(iHeight)
	, m_rSigmaX(rSigmaX)
	, m_rSigmaY(rSigmaY)
	, m_rWxl(rWxl)
	, m_rWxi(rWxi)
	, m_rWxh(rWxh)
	, m_rWyl(rWxl)
	, m_rWyi(rWyi)
	, m_rWyh(rWyh)
	, m_rThr(rThr)
	, m_d_rThr(nullptr)
	, m_d_fPadded(nullptr)
	, m_d_xf(nullptr)
	, m_d_yf(nullptr)
	, im_d_Fg(nullptr)
	, im_d_filtered(nullptr)
	, m_fCoreTime(0)
	, m_fMemTime(0)
	, m_fPrecomputeTime(0)
{
	// Get the number of SMs on GPU 
	hipDeviceGetAttribute(&m_iSMs, hipDeviceAttributeMultiprocessorCount, 0);

	// scale for window so that norm2 of the window is 1. 
	m_rGaussianNorm2 = sqrt(4 * float(M_PI)*m_rSigmaX*m_rSigmaY);


	/* Do the Initialization */
	if (-1 == Initialize(z))
	{
		std::cout << "FFT padding is out of range [4096]. Shrink the size of either the image or the Gaussian Window!" << std::endl;
		throw - 1;
	}
}

WFF2_CUDAF::~WFF2_CUDAF()
{
	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);

	hipEventRecord(start);
	cudaSafeFree(m_d_fPadded);
	cudaSafeFree(m_d_xf);
	cudaSafeFree(m_d_yf);

	cudaSafeFree(im_d_filtered);
	cudaSafeFree(im_d_Fg);
	cudaSafeFree(im_d_Sf);
	cudaSafeFree(m_d_rThr);
	hipEventRecord(end);
	hipEventSynchronize(end);
	float t = 0;
	hipEventElapsedTime(&t, start, end);
	m_fMemTime += t;

	std::cout << "paWFF Memory alloc/dealloc time is: " << m_fMemTime << " ms" << std::endl;

	hipfftDestroy(m_planPadded);
}

void WFF2_CUDAF::operator()(hipfftComplex *d_f,
	WFT2_DeviceResultsF &d_z,
	double &time)
{
	cuWFF2(d_f, d_z, time);
}

void WFF2_CUDAF::cuWFF2(hipfftComplex *d_f, WFT2_DeviceResultsF &d_z, double &time)
{
	m_fCoreTime = 0;

	/* CUDA blocks & threads scheduling */
	dim3 threads(BLOCK_SIZE_16, BLOCK_SIZE_16);
	dim3 blocksPadded((m_iPaddedWidth + BLOCK_SIZE_16 - 1) / BLOCK_SIZE_16, (m_iPaddedHeight + BLOCK_SIZE_16 - 1) / BLOCK_SIZE_16);
	dim3 blocksImg((m_iWidth + BLOCK_SIZE_16 - 1) / BLOCK_SIZE_16, (m_iHeight + BLOCK_SIZE_16 - 1) / BLOCK_SIZE_16);
	int blocks1D = std::min((m_iPaddedWidth*m_iPaddedHeight + BLOCK_SIZE_256 - 1) / BLOCK_SIZE_256, 2048);

	hipEvent_t start, pre, end;
	hipEventCreate(&start);
	hipEventCreate(&pre);
	hipEventCreate(&end);
	/* Set the threshold m_rThr if it's not specified by the client */

	hipEventRecord(start);
	cuWFF2_SetThreashold(d_f);

	/* Feed the f to its padded version */
	cuWFF2_feed_fPadded(d_f);

	/* Pre-compute the FFT of m_d_fPadded */
	checkCudaErrors(hipfftExecC2C(m_planPadded, m_d_fPadded, m_d_fPadded, HIPFFT_FORWARD));

	/* Clear the results if they already contain last results */
	init_WFF_matrices_kernel << <blocksImg, threads >> >(d_z.m_d_filtered, m_iWidth, m_iHeight);
	getLastCudaError("init_WFF_matrices_kernel Launch Failed!");
	hipEventRecord(pre);


	/* Insert this part inbetween to realize kind of CPU&GPU concurrent execution.
	map the wl: wi : wh interval to integers from  0 to size = (wyh - wyl)/wyi + 1 in order to divide the
	copmutations across threads, since threads indices are more conviniently controlled by integers 	    */
	int iwx = int((m_rWxh - m_rWxl)*(1 / m_rWxi)) + 1;
	int iwy = int((m_rWyh - m_rWyl)*(1 / m_rWyi)) + 1;

	init_WFF_matrices_kernel <<<blocksPadded, threads>>>(im_d_filtered, m_iPaddedWidth, m_iPaddedHeight);
	getLastCudaError("init_WFF_matrices_kernel Launch Failed!");


	for (int y = 0; y < iwy; y++)
	{
		// Now we have equal number of kernels executed in each stream
		for (int x = 0; x < iwx; x++)
		{			
			// Construct Fg
			compute_Fg_kernel <<<blocks1D, BLOCK_SIZE_256>> > (
				m_d_xf, m_d_yf, m_iPaddedWidth, m_iPaddedHeight,
				x, y, m_rWxi, m_rWyi, m_rWxl, m_rWyl,
				m_rSigmaX, m_rSigmaY, m_rGaussianNorm2, im_d_Fg);
			getLastCudaError("compute_Fg_kernel Launch Failed!");

			// Compute sf=ifft2(Ff.*Fg)
			complex_pointwise_multiplication_kernel <<<blocks1D, BLOCK_SIZE_256>> > (
				m_d_fPadded, im_d_Fg, m_iPaddedHeight*m_iPaddedWidth, im_d_Sf);
			getLastCudaError("complex_pointwise_multiplication_kernel Launch Failed!");
			checkCudaErrors(hipfftExecC2C(m_planPadded, im_d_Sf, im_d_Sf, HIPFFT_BACKWARD));

			// Threshold the sf: sf=sf.*(abs(sf)>=thr); 
			threshold_sf_kernel <<<blocksPadded, threads>>> (im_d_Sf, m_iWidth, m_iHeight, m_iPaddedWidth, m_iPaddedHeight, m_rThr);
			getLastCudaError("threshold_sf_kernel Launch Failed!");

			// implement of IWFT: conv2(sf,w);
			checkCudaErrors(hipfftExecC2C(m_planPadded, im_d_Sf, im_d_Sf, HIPFFT_FORWARD));
			complex_pointwise_multiplication_kernel <<<blocks1D, BLOCK_SIZE_256 >>> (
				im_d_Sf, im_d_Fg, m_iPaddedHeight*m_iPaddedWidth, im_d_Sf);
			getLastCudaError("complex_pointwise_multiplication_kernel Launch Failed!");
			checkCudaErrors(hipfftExecC2C(m_planPadded, im_d_Sf, im_d_Sf, HIPFFT_BACKWARD));

			// Update partial results im_d_filtered
			update_WFF_partial_filtered_kernel <<<blocksImg, threads >>> (im_d_Sf, m_iWidth, m_iHeight, m_iPaddedWidth, m_iPaddedHeight, im_d_filtered);
			getLastCudaError("update_WFF_partial_filtered_kernel Launch Failed!");
		}
		
	}

	update_WFF_final_filtered_kernel << <blocks1D, BLOCK_SIZE_256 >> >(im_d_filtered, m_iWidth*m_iHeight, d_z.m_d_filtered);
	getLastCudaError("update_WFF_final_filtered_kernel Launch Failed!");

	scale_WFF_final_filtered_kernel << <blocks1D, BLOCK_SIZE_256 >> >(d_z.m_d_filtered, m_iWidth*m_iHeight, m_rWxi, m_rWyi);
	getLastCudaError("scale_WFF_final_filtered_kernel Launch Failed!");

	hipEventRecord(end);
	hipEventSynchronize(end);

	// Calculate the running time
	float t_precompute = 0;

	hipEventElapsedTime(&t_precompute, start, pre);
	hipEventElapsedTime(&m_fCoreTime, pre, end);

	m_fPrecomputeTime += t_precompute;

	hipEventDestroy(start);
	hipEventDestroy(pre);
	hipEventDestroy(end);

	std::cout << "paWFF Precomputation Time: " << m_fPrecomputeTime << " ms" << std::endl;
	time = double(m_fCoreTime);
}

/* Private functions */
int WFF2_CUDAF::Initialize(WFT2_DeviceResultsF &d_z)
{
	hipEvent_t start, ma, plan, end;
	hipEventCreate(&start);
	hipEventCreate(&ma);
	hipEventCreate(&plan);
	hipEventCreate(&end);

	/*----------------------------WFF&WFR Common parameters initialization-----------------------------*/
	// Half of the Gaussian Window size
	m_iSx = int(round(3 * m_rSigmaX));
	m_iSy = int(round(3 * m_rSigmaY));
	// Guassian Window Size
	m_iWinHeight = 2 * m_iSy + 1;
	m_iWinWidth = 2 * m_iSx + 1;

	// Calculate the initial padding in order to perform the cyclic convolution using FFT
	// The padding size is size(A) + size(B) - 1;
	m_iPaddedHeight = m_iHeight + m_iWinHeight - 1;
	m_iPaddedWidth = m_iWidth + m_iWinWidth - 1;

	// Calculate the second padding in order to fit the optimized size for FFT
	int iH = getFirstGreater(m_iPaddedHeight);
	int iW = getFirstGreater(m_iPaddedWidth);
	if (-1 == iH || -1 == iW)
	{
		// Out of range
		return -1;
	}
	else
	{
		m_iPaddedHeight = OPT_FFT_SIZE[iH];
		m_iPaddedWidth = OPT_FFT_SIZE[iW];

		int iPaddedSize = m_iPaddedHeight * m_iPaddedWidth;

		hipEventRecord(start);
		/* Memory Preallocation on Device */
		// Allocate memory for input padded f which is pre-copmuted and remain unchanged
		checkCudaErrors(hipMalloc((void**)&m_d_fPadded, sizeof(hipfftComplex)*iPaddedSize));
		checkCudaErrors(hipMalloc((void**)&m_d_xf, sizeof(hipfftReal)*iPaddedSize));
		checkCudaErrors(hipMalloc((void**)&m_d_yf, sizeof(hipfftReal)*iPaddedSize));
		hipEventRecord(ma);

		/* Make the CUFFT plans */
		checkCudaErrors(hipfftPlan2d(&m_planPadded, m_iPaddedHeight, m_iPaddedWidth, HIPFFT_C2C));

		hipEventRecord(plan);
		/* Construct the xf & yf */
		dim3 threads(BLOCK_SIZE_16, BLOCK_SIZE_16);
		dim3 blocks((m_iPaddedWidth + BLOCK_SIZE_16 - 1) / BLOCK_SIZE_16, (m_iPaddedHeight + BLOCK_SIZE_16 - 1) / BLOCK_SIZE_16);
		// Generate xf, yf
		gen_xf_yf_Kernel << <blocks, threads >> >(m_d_xf, m_d_yf, m_iPaddedWidth, m_iPaddedHeight);
		getLastCudaError("gen_xf_yf_Kernel Launch Failed!");

		// Shift xf, yf to match the FFT's results
		fftshift_xf_yf_kernel << <blocks, threads >> >(m_d_xf, m_d_yf, m_iPaddedWidth, m_iPaddedHeight);
		getLastCudaError("fftshift_xf_yf_kernel Launch Failed!");
		hipEventRecord(end);
		hipEventSynchronize(end);

		float f_time_MemAlloc = 0;
		float f_time_Pre = 0;
		
		hipEventElapsedTime(&f_time_MemAlloc, start, ma);
		hipEventElapsedTime(&f_time_Pre, plan, end);

		m_fMemTime += f_time_MemAlloc;
		m_fPrecomputeTime += f_time_Pre;

		cuWFF2_Init(d_z);
	}

	hipEventDestroy(start);
	hipEventDestroy(ma);
	hipEventDestroy(plan);
	hipEventDestroy(end);

	return 0;
}

void WFF2_CUDAF::cuWFF2_Init(WFT2_DeviceResultsF &d_z)
{
	int iImageSize = m_iWidth * m_iHeight;
	int iPaddedSize = m_iPaddedHeight * m_iPaddedWidth;

	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);

	hipEventRecord(start);
	// Allocate memory for the final results
	checkCudaErrors(hipMalloc((void**)&d_z.m_d_filtered, sizeof(hipfftComplex)*iImageSize));

	// 1. Allocate memory for intermediate results
	checkCudaErrors(hipMalloc((void**)&im_d_Fg, sizeof(hipfftComplex)*iPaddedSize));
	checkCudaErrors(hipMalloc((void**)&im_d_filtered, sizeof(hipfftComplex)*iPaddedSize));
	checkCudaErrors(hipMalloc((void**)&im_d_Sf, sizeof(hipfftComplex)*iPaddedSize));
	
	// 2. Calculate threshold
	if (m_rThr <= 1e-6)
	{
		checkCudaErrors(hipMalloc((void**)&m_d_rThr, sizeof(float)));
	}
	hipEventRecord(end);
	hipEventSynchronize(end);

	float time = 0;

	hipEventElapsedTime(&time, start, end);

	m_fMemTime += time;

	hipEventDestroy(start);
	hipEventDestroy(end);
}

void WFF2_CUDAF::cuWFF2_feed_fPadded(hipfftComplex *d_f)
{
	dim3 threads(BLOCK_SIZE_16, BLOCK_SIZE_16);
	dim3 blocks((m_iPaddedWidth + BLOCK_SIZE_16 - 1) / BLOCK_SIZE_16, (m_iPaddedHeight + BLOCK_SIZE_16 - 1) / BLOCK_SIZE_16);

	feed_fPadded_kernel <<<blocks, threads>>>(d_f, m_d_fPadded, m_iWidth, m_iHeight, m_iPaddedWidth, m_iPaddedHeight);
	getLastCudaError("feed_fPadded_kernel Launch Failed!");
}

void WFF2_CUDAF::cuWFF2_SetThreashold(hipfftComplex *d_f)
{
	// Set the m_rThr if not set
	if (m_rThr < 0)
	{
		int iImgSize = m_iWidth * m_iHeight;

		// Launch the kernel to compute the threshold
		int blocks = std::min((iImgSize + BLOCK_SIZE_256 - 1) / BLOCK_SIZE_256, 2048);
		compute_WFF_threshold_kernel << <blocks, BLOCK_SIZE_256 >> >(d_f, m_d_rThr, iImgSize);
		getLastCudaError("compute_WFF_threshold_kernel Launch Failed!");

		// Passing back to host
		checkCudaErrors(hipMemcpy(&m_rThr, m_d_rThr, sizeof(float), hipMemcpyDeviceToHost));
		m_rThr = 6 * sqrt(m_rThr *(1.0f / float(iImgSize)) / 3.0f);
	}
}