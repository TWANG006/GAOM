#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<>

#include "vec_add.cuh"
#include <iostream>

using namespace std;

__global__
void vecAdd_kernel(float *d_A, float *d_B, float *d_C, int n)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < n)
		d_C[i] = d_A[i] + d_B[i];
}

void vecAdd(float* h_A, float *h_B, float *h_C, int n)
{
	int size = n * sizeof(float);
	
	float *d_A, *d_B, *d_C;	// device pointers
	hipEvent_t start, end, ma, hd, ke, dh;	// cuda events used for timing
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventCreate(&ma);
	hipEventCreate(&hd);
	hipEventCreate(&ke);	
	hipEventCreate(&dh);

	hipEventRecord(start);

	// Device memory allocation
	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);
	hipMalloc((void**)&d_C, size);

	hipEventRecord(ma);

	// Host->device memory transfer
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	hipEventRecord(hd);

	// Kernel launch
	vecAdd_kernel<<<(n + 256 - 1) / 256, 256>>>(d_A, d_B, d_C, n);

	hipEventRecord(ke);

	// Device->Host memory transfer
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	hipEventRecord(dh);

	// Device memory deallocation
	hipFree(d_A);	hipFree(d_B);	hipFree(d_C);

	hipEventRecord(end);
	hipEventSynchronize(end);

	float ma_time = 0, hd_time = 0, ke_time = 0, dh_time = 0, md_time = 0, total_time = 0;
	hipEventElapsedTime(&ma_time, start, ma);
	hipEventElapsedTime(&hd_time, ma, hd);
	hipEventElapsedTime(&ke_time, hd, ke);
	hipEventElapsedTime(&dh_time, ke, dh);
	hipEventElapsedTime(&md_time, dh, end);
	hipEventElapsedTime(&total_time, start, end);

	cout << "CUDA Vector Addition overall running time: " << total_time << " ms" << endl;
	cout << "Memory allocation time: " << ma_time << " ms" << endl;
	cout << "Memory transferring time: " << hd_time + dh_time << " ms" << endl;
	cout << "Kernel execution time: " << ke_time << " ms" << endl;
	cout << "Memory deallocation time: " << md_time << " ms" << endl;
}